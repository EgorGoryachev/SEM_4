#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <cstdio>
#include <stdlib.h>
#define N (33 * 1024)


__global__ void add(int* a, int* b, int* c)
{
    int tid = blockIdx.x * blockDim.x + threadIdx.x;
    if (tid < N)
    {
        c[tid] = a[tid] + b[tid];
    }
}

int main(void)
{
    int* a, * b, * c;
    int* dev_a, * dev_b, * dev_c;
    a = (int*)malloc(N * sizeof(int));
    b = (int*)malloc(N * sizeof(int));
    c = (int*)malloc(N * sizeof(int));
    hipMalloc((void**)&dev_a, N * sizeof(int));
    hipMalloc((void**)&dev_b, N * sizeof(int));
    hipMalloc((void**)&dev_c, N * sizeof(int));
    for (int i = 0; i < N; i++)
    {
        a[i] = i;
        b[i] = 2 * i;
    }
    hipMemcpy(dev_a, a, N * sizeof(int), hipMemcpyHostToDevice);
    hipMemcpy(dev_b, b, N * sizeof(int), hipMemcpyHostToDevice);

    add << < (N + 255) / 256, 256) >> > (dev_a, dev_b, dev_c);

    hipMemcpy(c, dev_c, N * sizeof(int), hipMemcpyDeviceToHost);
    bool success = true;
    for (int i = 0; i < N; i++)
    {
        if ((a[i] + b[i]) != c[i])
        {
            printf("Error:  %d + %d != %d\n", a[i], b[i], c[i]);
            success = false;
        }
    }
    if (success) {
        printf("The work is completed\n");
    }

    hipFree(dev_a);
    hipFree(dev_b);
    hipFree(dev_c);

    free(a);
    free(b);
    free(c);
    return 0;
}